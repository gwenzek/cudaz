
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif
__global__ void rgba_to_greyscale(const uchar3 *const rgbaImage,
                                  unsigned char *const greyImage, int numRows,
                                  int numCols) {
  // TODO
  // Fill in the kernel to convert from color to greyscale
  // the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  // The output (greyImage) at each pixel should be the result of
  // applying the formula: output = .299f * R + .587f * G + .114f * B;
  // Note: We will be ignoring the alpha channel for this conversion

  // First create a mapping from the 2D block and grid locations
  // to an absolute 2D location in the image, then use that to
  // calculate a 1D offset
  uint myId = blockIdx.x * blockDim.x + threadIdx.x;
  if (myId >= numRows * numCols) return;
  uchar3 px = rgbaImage[myId];
  float R = px.x;
  float G = px.y;
  float B = px.z;
  float output = (0.299f * R + 0.587f * G + 0.114f * B);
  greyImage[myId] = (unsigned char)output;
}

#ifdef __cplusplus
}
#endif
