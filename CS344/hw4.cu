#include "hip/hip_runtime.h"
#include <math.h>
#include "cuda_helpers.h"

#ifdef __cplusplus
extern "C" {
#endif

const uint _radix_mask = (2 << 4) - 1;

__device__ uchar readChannel(const uchar3* d_pixels, int offset, uint channel) {
    uchar* channel_ptr = (uchar*)d_pixels;
    return channel_ptr[3 * offset + channel];
}

__device__ void writeChannel(uchar3* d_pixels, int offset, uint channel, uchar val) {
    uchar* channel_ptr = (uchar*)d_pixels;
    channel_ptr[3 * offset + channel] = val;
}

__global__ void sort_network(const float* d_in) {
    uint i = threadIdx.x;
    uint n = threadDim.x;
    SHARED(d_group, uint);
    d_group[i] = d_in[ID_X];
    __syncthreads();

    // Using notation from https://en.wikipedia.org/wiki/Bitonic_sorter#How_the_algorithm_works
    for (uint k = 2; k <= n; k *= 2) {
        // Corresponds to green blocks
        bool upward = (i & k) == 0;
        for (uint j = k/2; j >=0; j/=2) {
            uint l = i ^ j;
            if (l > i) {
                bool sorted = arr[i] < arr[l];
                if ((upward & !sorted) || (!upward & sorted)) {
                    float arr_i = arr[i];
                    arr[i] = arr[l];
                    arr[l] = arr_i;
                }
            }
            __syncthreads();
        }
    }
    d_in[ID_X] = d_group[i];
}

__global__ void find_radix(
    const float* d_in,
    uchar* d_out,
    uchar shift,
    int n
) {
    uint id = ID_X;
    if (id >= n) return;
    uint x = ((uint*)d_in)[id];
    x = x >> shift & _radix_mask;
    d_out[id] = (uchar)x;
}

__global__
void radix_cdf(const uchar* d_radix, uint* h, int n) {
    int tid = ID_X;
    if (tid >= n) return;
    int group = blockIdx.y;
    SHARED(d_bins, uint);
    d_bins[tid] = d_radix[tid] == (uchar)group ? 1 : 0;
    __syncthreads();
    // Reduce
    uint step = 1;
    for (; step < n; step *= 2) {
        if (tid >= step && (n - 1 - tid) % (step * 2) == 0) {
            d_bins[tid] += d_bins[tid - step];
        }
        __syncthreads();
    }

    if (tid == n - 1) d_bins[tid] = 0;

    // Downsweep
    for (step /= 2; step > 0; step /= 2) {
        if (tid >= step && (n - 1 - tid) % (step * 2) == 0) {
            uint left = d_bins[tid - step];
            uint right = d_bins[tid];
            d_bins[tid] = left + right;
            d_bins[tid - step] = right;
        }
        __syncthreads();
    }
    d_bins_out[blockIdx.y * n + ID_X] = d_bins[tid];
}

__global__
void cdf_vertical(uint* d_cdf, int n, int step) {
    int  tid = ID_X;
    if (tid >= n) return;
    // Here thread id correspond to a grid id in radix_cdf
    int group = tid;
    SHARED(d_bins, uint);
    d_bins[tid] = d_cdf[(group + 1) * step - 1];
    // Reduce
    uint step = 1;
    for (; step < n; step *= 2) {
        if (tid >= step && (n - 1 - tid) % (step * 2) == 0) {
            d_bins[tid] += d_bins[tid - step];
        }
        __syncthreads();
    }

    if (tid == n - 1) d_bins[tid] = 0;

    // Downsweep
    for (step /= 2; step > 0; step /= 2) {
        if (tid >= step && (n - 1 - tid) % (step * 2) == 0) {
            uint left = d_bins[tid - step];
            uint right = d_bins[tid];
            d_bins[tid] = left + right;
            d_bins[tid - step] = right;
        }
        __syncthreads();
    }
    d_cdf[group * step] += d_bins[tid];
}

__global__
void cdf_increment_after_vertical(uint* d_cdf, int n, int step) {
    uint tid = ID_X;
    uint group_start = (tid % step) * step;
    if (tid == group_start || tid >= n) return;

    d_cdf[tid] += d_cdf[group_start];
}

__global__ void naive_normalized_cross_correlation(
    float *d_response, uchar3 *d_original,
    uchar3 *d_template,
    int num_pixels_y,
    int num_pixels_x,
    int template_half_height,
    int template_height,
    int template_half_width,
    int template_width,
    int template_size
) {
  int ny = num_pixels_y;
  int nx = num_pixels_x;
  int knx = template_width;
  int2 image_index_2d = {
    (int)((blockIdx.x * blockDim.x) + threadIdx.x),
    (int)((blockIdx.y * blockDim.y) + threadIdx.y)
  };
  int channel = threadIdx.z;
  int image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

  if (image_index_2d.x < nx && image_index_2d.y < ny) {
    // compute image mean
    float image_sum = 0.0f;
    float template_sum = 0.0f;

    for (int y = -template_half_height; y <= template_half_height; y++) {
      for (int x = -template_half_width; x <= template_half_width; x++) {
        int2 image_offset_index_2d = {
            CLAMP(image_index_2d.x + x, nx),
            CLAMP(image_index_2d.y + y, ny)
        };
        int image_offset_index_1d = (nx * image_offset_index_2d.y) + image_offset_index_2d.x;

        uchar original = readChannel(d_original, image_offset_index_1d, channel);
        image_sum += (float)original;
      }
    }

    float template_mean = template_sum / (float)template_size;
    float image_mean = image_sum / (float)template_size;

    // compute sums
    float sum_of_image_template_diff_products = 0.0f;
    float sum_of_squared_image_diffs = 0.0f;
    float sum_of_squared_template_diffs = 0.0f;

    for (int y = -template_half_height; y <= template_half_height; y++) {
      for (int x = -template_half_width; x <= template_half_width; x++) {
        int2 image_offset_index_2d = {
            CLAMP(image_index_2d.x + x, nx),
            CLAMP(image_index_2d.y + y, ny)
        };
        int image_offset_index_1d =
            (nx * image_offset_index_2d.y) +
            image_offset_index_2d.x;

        unsigned char image_offset_value = readChannel(d_original, image_offset_index_1d, channel);
        float image_diff = (float)image_offset_value - image_mean;

        int2 template_index_2d = {x + template_half_width, y + template_half_height};
        int template_index_1d = (knx * template_index_2d.y) + template_index_2d.x;

        unsigned char template_value = readChannel(d_template, template_index_1d, channel);
        float template_diff = template_value - template_mean;

        float image_template_diff_product = image_offset_value * template_diff;
        float squared_image_diff = image_diff * image_diff;
        float squared_template_diff = template_diff * template_diff;

        sum_of_image_template_diff_products += image_template_diff_product;
        sum_of_squared_image_diffs += squared_image_diff;
        sum_of_squared_template_diffs += squared_template_diff;
      }
    }

    //
    // compute final result
    //
    float result_value = 0.0f;

    if (sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0) {
      result_value =
          sum_of_image_template_diff_products /
          sqrt(sum_of_squared_image_diffs * sum_of_squared_template_diffs);
    }

    d_response[image_index_1d] = result_value;
  }
}

__global__
void reduce_min(const float* d_in, float* d_out, int num_pixels)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    SHARED(sdata, float);
    int id = ID_X;
    if (id >= num_pixels) {
      return;
    }
    int tid  = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[id];
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = MIN(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__
void range(uint* d_out, uint n) {
    int id = ID_X;
    if (id >= n) return;
    d_out[id] = id;
}

__global__ void remove_redness(
    const unsigned int *d_coordinates, const uchar3 *d_rgb, uchar3 *d_out,
    int num_coordinates,
    int num_pixels_y, int num_pixels_x,
    int template_half_height, int template_half_width
) {
  int ny = num_pixels_y;
  int nx = num_pixels_x;
  int global_index_1d = ID_X;

  int imgSize = num_pixels_x * num_pixels_y;

  if (global_index_1d < num_coordinates) {
    uint image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
    uint2 image_index_2d = {
        image_index_1d % num_pixels_x,
        image_index_1d / num_pixels_x
    };

    for (int y = image_index_2d.y - template_half_height;
         y <= image_index_2d.y + template_half_height; y++) {
      for (int x = image_index_2d.x - template_half_width;
           x <= image_index_2d.x + template_half_width; x++) {
        int clamped_index = (nx * CLAMP(y, ny)) + CLAMP(x, nx);

        uchar g_value = readChannel(d_rgb, clamped_index, 1);
        uchar b_value = readChannel(d_rgb, clamped_index, 2);
        uchar gb_average = ((uint)g_value + (uint)b_value) / 2;

        writeChannel(d_out, clamped_index, 0, gb_average);
      }
    }
  }
}

#ifdef __cplusplus
}
#endif
